
#include <hip/hip_runtime.h>
__global__ void per_row_kernel(int m,int n,int *A,int *B,int *C){
    int idr;
    idr=blockIdx.x*blockDim.x+threadIdx.x;
    if(idr<m){
          for(int j=0;j<n;j++){
            C[idr*n+j]=A[idr*n+j]+B[idr*n+j];
        }  
    }
    
}


__global__ void per_column_kernel(int m,int n,int *A,int *B,int *C){
    int idc;
    idc =blockDim.x*blockDim.y*blockIdx.x+ blockDim.x* threadIdx.y+threadIdx.x;
    if(idc<n){
          for(int j=0;j<m;j++){
            C[idc*m+j]=A[idc*m+j]+B[idc*m+j];
        }  
    }
    
}


__global__ void per_element_kernel(int m,int n,int *A,int *B,int *C){
    int i,j,ide;
    i = threadIdx.x + blockDim.x * threadIdx.y;
    j = blockIdx.x + gridDim.x * blockIdx.y;
    ide=j*blockDim.x*blockDim.y+i;
    if(ide<(m*n))
    C[ide]=A[ide]+B[ide];
    
}