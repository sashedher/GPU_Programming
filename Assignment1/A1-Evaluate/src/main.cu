#include "hip/hip_runtime.h"
/*
author : Anshul Farkya
created on : 12-feb-2021
*/
#include <stdio.h>
#include <stdlib.h>
#include "kernels.h"


// Print the matrix
void printMatrix(int *mat, int rows, int cols)
{
  for(int i=0; i < rows; i++) 
  {
    for(int j=0; j < cols; j++) 
	{ 
      printf("%d ", mat[i*cols+j]); 
	}
    printf("\n");
  } 
  printf("\n");
}

// Verify the correctness by comparing the sequential output with parallel output
bool compareMatrices(int *gpu, int *ref, int rows, int cols)
{

		for(int i=0; i < rows; i++)
		{
				for(int j=0; j < cols; j++)
				{
						if (ref[i*cols + j] != gpu[i*cols +j])
						{
								return false;
						}
				}
		}
		return true; // generated output matches expected output
}

// Generating the expected output 
void add_matrices(int A[], int B[], int out[], int rows, int cols)
{
  for(int i=0; i<rows; i++)
  {
	for(int j=0; j<cols; j++)
	{
        	out[i*cols + j] = A[i*cols + j] + B[i*cols + j];
	}
  }
}


int main(int argc, char** argv)
{
	FILE *fp;
	fp = fopen(argv[1], "r");
	if(fp == NULL)
	{
			printf("Failed to open input file %s\n", argv[1]);
			exit(0);
	}
	
	// Reading the input matrix from file
	int rows,cols; // holds the size of the input square matrix
	fscanf(fp, "%d", &rows);
	fscanf(fp, "%d", &cols);
	//printf("rows = %d\n",rows);
	//printf("cols = %d\n",cols);
	int numBytes = rows * cols * sizeof(int);
	
	long long k1_blkDimX, k1_gridDimX, k2_blkDimX, k2_blkDimY, k2_gridDimX, k3_blkDimX, k3_blkDimY, k3_gridDimX, k3_gridDimY;
	k1_blkDimX = 512;
	k1_gridDimX = ceil(float(rows*cols)/k1_blkDimX);
	//printf("k1_gridDimX = %d\n",k1_gridDimX);
	int *A = (int*) malloc(numBytes);
	for(int i = 0; i < rows*cols; i++)
	{
			fscanf(fp, "%d", &A[i]);
	}
	
	int *B = (int*) malloc(numBytes);
	for(int i = 0; i < rows*cols; i++)
	{
			fscanf(fp, "%d", &B[i]);
	}

	// computing the addition on CPU to generate expected output
	int *cpuOut = (int*) malloc(numBytes);
	add_matrices(A, B, cpuOut, rows, cols);
	//printMatrix(cpuOut, rows,cols);

	int *d_A;
	hipMalloc(&d_A, numBytes);
	hipMemcpy(d_A, A, numBytes, hipMemcpyHostToDevice);
	
	int *d_B;
	hipMalloc(&d_B, numBytes);
	hipMemcpy(d_B, B, numBytes, hipMemcpyHostToDevice);
	
	int *d_C;
	hipMalloc(&d_C, numBytes);
	
	// Launching the first kernel
	per_row_kernel<<<k1_gridDimX, k1_blkDimX>>>(rows,cols,d_A, d_B, d_C);
	hipDeviceSynchronize();
	// Copying back from the device
	int *out1 = (int*) malloc(numBytes);
	hipMemcpy(out1, d_C, numBytes, hipMemcpyDeviceToHost);
	//printMatrix(out1, rows,cols);
	printf("%s,%s\n", argv[1], compareMatrices(out1, cpuOut, rows, cols) ? "Success" : "Failure");
	
	hipMemset(d_C, 0, numBytes);
	k2_blkDimX = 32;
	k2_blkDimY = 32;
	k2_gridDimX = ceil(float(rows*cols)/(k2_blkDimX*k2_blkDimY));
	// Launching the second kernel
	dim3 block2D(k2_blkDimX, k2_blkDimY,1);
	per_column_kernel<<<k2_gridDimX, block2D>>>(rows,cols,d_A, d_B, d_C);
	hipDeviceSynchronize();
	// Copying back from the device
	int *out2 = (int*) malloc(numBytes);
	hipMemcpy(out2, d_C, numBytes, hipMemcpyDeviceToHost);
  //printMatrix(out2, rows,cols);
  printf("%s,%s\n", argv[1],compareMatrices(out2, cpuOut, rows, cols) ? "Success" : "Failure");

	hipMemset(d_C, 0, numBytes);
	k3_blkDimX = 32;
	k3_blkDimY = 32;
	k3_gridDimX = 32;
	k3_gridDimY = ceil(float(rows*cols)/(k3_blkDimX*k3_blkDimY*k3_gridDimX));
	
	// Launching the third kernel
	dim3 grid2D(k3_gridDimX, k3_gridDimY);
	dim3 blk2D(k3_blkDimX,k3_blkDimY);
	per_element_kernel<<<grid2D, blk2D>>>(rows,cols,d_A, d_B, d_C);
	hipDeviceSynchronize();
	// Copying back from the device
	int *out3 = (int*) malloc(numBytes);
	hipMemcpy(out3, d_C, numBytes, hipMemcpyDeviceToHost);
	printMatrix(out3, rows,cols);
	printf("%s,%s\n", argv[1],compareMatrices(out3, cpuOut, rows,cols) ? "Success" : "Failure");

	// Deallocating the memory
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	free(cpuOut);
	free(out1);
	free(out2);
	free(out3);

	return(0);

}
